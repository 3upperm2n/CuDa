#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


// data type: float
// 3 channels
// hipReadModeElementType:	Read texture as specified element type
// hipReadModeNormalizedFloat: Read texture as normalized float
texture<float, 3, hipReadModeElementType> tex3d;


__global__ void test_kernel(const int w, const int h, const int d) 
{
    float data;

    //data = tex3D(tex3d, 0.f, 0.f, (float) threadIdx.x);
	//printf("thread %d, data %f\n", threadIdx.x, data);

    data = tex3D(tex3d, 0.f, 2.f, (float) threadIdx.x);
	printf("thread %d, data %f\n", threadIdx.x, data);
}

//----------------------------------------------------------------------------//
// main
//----------------------------------------------------------------------------//
int main(void)
{
	int w = 2;
	int h = 3;
	int d = 4;

	//-----------//
	// 3d array on the host 
	//-----------//
	float *h_array = NULL;
	h_array = (float*) malloc(w * h * d * sizeof(float));

	printf("\nInput array:\n\n");

	for (int i=0; i<w; i++) {
		for (int j=0; j<h; j++) {
			for (int k=0; k<d; k++) {
				h_array[k*w*h + j*w + i] = (float)(i + j + k);
				printf("%12.6f ", (float)(i + j + k));
			}
			printf("\n");
		}
		printf("\n");
	}
	printf("\n");

	//-----------//
	// 3d array on device
	//-----------//
	// set up the cuda array
	hipArray *d_array= NULL;
	hipChannelFormatDesc chanDesc = hipCreateChannelDesc<float>();
	hipExtent const array_dim = {w, h, d};
	checkCudaErrors(hipMalloc3DArray(&d_array, &chanDesc, array_dim));


	// paramters
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.extent   = array_dim;
	copyParams.srcPtr   = make_hipPitchedPtr((void *)h_array, 
			array_dim.width*sizeof(float), array_dim.width, array_dim.height);
	copyParams.dstArray = d_array;
	copyParams.kind     = hipMemcpyHostToDevice;
	
	//---------------//
	// copy data from host to device
	//---------------//
	checkCudaErrors(hipMemcpy3D(&copyParams));

	//---------------//
	// binding array to texture
	//---------------//
	// set texture parameters
	// http://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#axzz4Xfram8Ds
	tex3d.normalized = false;
	tex3d.filterMode = hipFilterModePoint;
	tex3d.addressMode[0] = hipAddressModeWrap;   // wrap texture coordinates
	tex3d.addressMode[1] = hipAddressModeWrap;
	tex3d.addressMode[2] = hipAddressModeWrap;

	// bind array to 3D texture
	checkCudaErrors(hipBindTextureToArray(tex3d, d_array, chanDesc));

	test_kernel <<< 1, 4 >>> (w,h,d);	

	if(h_array != NULL) free(h_array);

	if(d_array != NULL) hipFree(d_array);

	hipDeviceReset();

	return 0;
}
